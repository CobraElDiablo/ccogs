#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <math.h>
#include <assert.h>
#include <iostream.h>
#include <iomanip.h>
#include <fstream.h>

#include <unistd.h>



#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "cutil.h"


///////////////////////////////////// this version calculates the aperture mass at the galaxy positions. 



void checkCUDAerror(const char *msg);

int checkDeviceSpecs(int number_of_galaxies);



__global__ void mApKernel(float* rgamma1, float* rgamma2, float* ra, float* dec, float* mAp_rgamma, float* var_rgamma, float* SN_rgamma,  int tot_gals, float theta_max)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
   
  float this_ra = ra[idx];
  float this_dec = dec[idx];
  
  // want to include the tails outside the halo radius to which our filter is tuned......
  int kernel_radius = 1.5*theta_max;
  float ang = 0.0;
  float xc = 0.15; // could be passed to fn
  float x = 0, Q = 0;
  
  float rgammaMap = 0;
  float rgammaVar=0;
  float radiff, decdiff, dist;

  float npoints = 0;

  for(int i=0; i<tot_gals; i++){

    radiff = (float)this_ra-ra[i]; 
    if(abs(radiff)>kernel_radius) continue;
    decdiff=(float)this_dec-dec[i];
    if(abs(decdiff)>kernel_radius || (radiff==0 && decdiff==0)) continue;
    
    dist = sqrtf(radiff*radiff + decdiff*decdiff);
    if(abs(dist)>kernel_radius) continue;
    
    // have to do something a bit complicated for teh angle - make sure it's getting the correct range. 
    // based on Jan's get_angle(radiff, decdiff)
    if(radiff==0 && decdiff>0) ang = M_PI/2.0;
    else if(radiff==0 && decdiff<0) ang = -1.0 * M_PI/2.0;
    else if(radiff>0) ang = atanf(decdiff/radiff);
    else if(radiff<0 && decdiff>0) ang = atanf(decdiff/radiff) + M_PI;
    else if(radiff<0 and decdiff<0) ang = atanf(decdiff/radiff)-M_PI;
    
    x = dist / theta_max;
    Q = (1.0 / (1.0 + exp(6.0 - 150.0*x) + exp(-47.0 + 50.0*x))) * (tanh(x/xc) / (x/xc));
    
    rgammaMap+=Q* (-1* (rgamma1[i]*cos(2*ang) + rgamma2[i]*sin(2*ang) ));
    rgammaVar+= Q*Q* (rgamma1[i]*rgamma1[i] + rgamma2[i]*rgamma2[i]);
    
    
     npoints++;

}
  
  
  mAp_rgamma[idx] = rgammaMap/npoints;// got to normalise by the # gals I did the sum over. 
  var_rgamma[idx] = rgammaVar /(2*npoints*npoints); 
  SN_rgamma[idx] = sqrtf(2) * rgammaMap / sqrtf(rgammaVar);

}




////////////////////////////////////////////////////////////////////////////////////
//////////  ********** aperture mass for ellip ***************  ////////////////////
////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv){
  

  char* input_filename; char* output_filename;
  int number_of_galaxies; 
  float filter_rad; 
  if (argc>1)
    {
      input_filename = argv[1];
      output_filename = argv[2];
      number_of_galaxies = atoi(argv[3]);
      filter_rad = atof(argv[4]);
    }

 

     
  // CPU memory
  size_t sizeneeded = number_of_galaxies*sizeof(float);
  float *h_rgamma1 = 0;
  float *h_rgamma2 = 0;
  float *h_ra = 0;
  float *h_dec = 0;
  h_rgamma1 = (float*) malloc(sizeneeded);
  h_rgamma2 = (float*) malloc(sizeneeded);
  h_ra = (float*) malloc(sizeneeded);
  h_dec = (float*) malloc(sizeneeded);
  

  ifstream infile;
  infile.open(input_filename);
  
  int i=0;
  float x, y, g1, g2;
  while(1)
    {
      infile>>x>>y>>g1>>g2;
      h_ra[i] = x;
      h_dec[i] = y;
      h_rgamma1[i] = g1;
      h_rgamma2[i] = g2;   
      i += 1;
      if(!infile.good()) break;       

    }
             
    
  // check whether the device has the capacity to do this calculation. 
  int max_threads = checkDeviceSpecs(number_of_galaxies);

  
  /// first, I need to test whether the device is busy. If so, it can wait a little while.
    // of course, even when this loop finished the device may still be busy. So it goes. 
    while(1){
      size_t testsize = 1*sizeof(float); 
      float *d_test;
      hipMalloc(&d_test, testsize);
      hipError_t err = hipGetLastError();
      if( hipSuccess != err){
	printf("gotta wait for a bit!: %s\n",  hipGetErrorString( err) );
	sleep(10);
      }
      else break;
    }
    
    
    // GPU memory
    float *d_rgamma1, *d_rgamma2, *d_ra, *d_dec;
    hipMalloc(&d_rgamma1, sizeneeded);
    hipMalloc(&d_rgamma2, sizeneeded);
    hipMalloc(&d_ra, sizeneeded);
    hipMalloc(&d_dec, sizeneeded);
    
    // output_mAp vector is going to be the calculated value for each point
    size_t sizeneeded_out = number_of_galaxies*sizeof(float);
    float *h_mAp_rgamma,*d_mAp_rgamma, *h_var_rgamma, *d_var_rgamma, *h_SN_rgamma, *d_SN_rgamma;
    
    h_mAp_rgamma = (float*)malloc(sizeneeded_out);
    hipMalloc(&d_mAp_rgamma, sizeneeded_out);
    h_var_rgamma = (float*)malloc(sizeneeded_out);
    hipMalloc(&d_var_rgamma, sizeneeded_out);
    h_SN_rgamma = (float*)malloc(sizeneeded_out);
    hipMalloc(&d_SN_rgamma, sizeneeded_out);
    
    
    //copy vectors from host to device memory
    hipMemcpy(d_rgamma1, h_rgamma1, sizeneeded, hipMemcpyHostToDevice);
    hipMemcpy(d_rgamma2, h_rgamma2, sizeneeded, hipMemcpyHostToDevice);
    hipMemcpy(d_ra, h_ra, sizeneeded, hipMemcpyHostToDevice);
    hipMemcpy(d_dec, h_dec, sizeneeded, hipMemcpyHostToDevice);
    hipMemcpy(d_mAp_rgamma, h_mAp_rgamma, sizeneeded_out, hipMemcpyHostToDevice);
    hipMemcpy(d_var_rgamma, h_var_rgamma, sizeneeded_out, hipMemcpyHostToDevice);
    hipMemcpy(d_SN_rgamma, h_SN_rgamma, sizeneeded_out, hipMemcpyHostToDevice);
    
    //check memory is alright
    if (0==h_rgamma1 || 0==h_rgamma2  || 0==h_ra || 0==h_dec || 0==h_mAp_rgamma || 0==h_var_rgamma|| 0==h_SN_rgamma ) printf("can't allocate memory on host \n");
    if (0==d_rgamma1 || 0==d_rgamma2  || 0==d_ra || 0==d_dec  || 0==d_mAp_rgamma || 0==d_var_rgamma || 0==d_SN_rgamma ) printf("can't allocate memory on device \n");
    checkCUDAerror("memory");
    
    
    
    // set up kernel params
    int threadsPerBlock = max_threads; // 
    int blocksPerGrid = int(ceil(number_of_galaxies / float(max_threads)) ); // need nx*nx threads total
    printf(" theads per block: %d and blocks per grid: %d for a total of: %d\n", threadsPerBlock, blocksPerGrid, threadsPerBlock*blocksPerGrid);
    
    
    mApKernel<<<blocksPerGrid, threadsPerBlock >>>(d_rgamma1, d_rgamma2, d_ra, d_dec, d_mAp_rgamma, d_var_rgamma, d_SN_rgamma,  number_of_galaxies, filter_rad); 
    
    checkCUDAerror("kernel");
    
    
    //get the output_mAp back off the device
    hipMemcpy(h_mAp_rgamma, d_mAp_rgamma, sizeneeded_out, hipMemcpyDeviceToHost);
    hipMemcpy(h_var_rgamma, d_var_rgamma, sizeneeded_out, hipMemcpyDeviceToHost);
    hipMemcpy(h_SN_rgamma, d_SN_rgamma, sizeneeded_out, hipMemcpyDeviceToHost);
    
    
    
    // finally, write out to the output file! 
    
   
    FILE *output_file;
    i=0;
    double sq2=sqrt(2.0);
    output_file = fopen(output_filename, "w");
    fprintf(output_file, " # ra  dec  mAp  Var S/N \n");
    for(i=0 ; i<number_of_galaxies; i++){
      fprintf(output_file, "%f %f %f %f %f \n", h_ra[i], h_dec[i], h_mAp_rgamma[i], h_var_rgamma[i],  h_SN_rgamma[i]);
    }
    fclose(output_file);
    
    printf("successfuly completed!\n");
}





//////////////////////////////////////////////////////////////////////////////////////////////////
//simple function to check for errors. From Dr Dobbs. 
void checkCUDAerror(const char *msg)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) 
    {
      fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
	      hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}



//////////////////////////////////////////////////////////////////////////////////

int checkDeviceSpecs( int number_of_galaxies){

  /// what size simulation are we trying to use?

  /// in case we decide to calcuate aperture mass at more points than the number of galaxies...

  int nxny = number_of_galaxies;

  int gpu_mem_needed = int(number_of_galaxies * sizeof(float)) * 7; // need to allocate gamma1, gamma2, ra, dec and output mAp and var and SN. 
  printf("Requirements: %d calculations and %d bytes memory on the GPU \n\n", nxny, gpu_mem_needed);  

  int threadsPerBlock=0;
  // now get the info from the device. 
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  if (error_id != hipSuccess) {
    printf( "hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id) );
  }
  // This function call returns 0 if there are no CUDA capable devices.
  if (deviceCount == 0)
    printf("There is no device supporting CUDA\n");
  else
    printf("Found %d CUDA Capable device(s)\n", deviceCount); 
  
  
  int dev, driverVersion = 0, runtimeVersion = 0;     
  for (dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
    
    printf("  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n", 
	   (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
    
    
    printf("  Warp size:                                     %d\n", deviceProp.warpSize);
    printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
    printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
	   deviceProp.maxThreadsDim[0],
	   deviceProp.maxThreadsDim[1],
	   deviceProp.maxThreadsDim[2]);
    printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
	   deviceProp.maxGridSize[0],
	   deviceProp.maxGridSize[1],
	   deviceProp.maxGridSize[2]);
    
    
    
    // does this device have enough capcacity for the calculation? 
        
    // check memory
    if((unsigned long long) deviceProp.totalGlobalMem < gpu_mem_needed) {
      printf(" FAILURE: Not eneough memeory on device for this calculation! \n");
      exit(1);
    }    
    else
      { 
	printf("Hurrah! This device has enough memory to perform this calculation\n");
	
	// check # threads
	
	threadsPerBlock = deviceProp.maxThreadsPerBlock; // maximal efficiency exists if we use max # threads per block. 
	int blocksPerGrid = int(ceil(nxny / threadsPerBlock)); // need nx*nx threads total

	if( int(deviceProp.maxThreadsDim[1])*int(deviceProp.maxThreadsDim[2]) <blocksPerGrid) {
	  printf("FAILURE: Not enough threads on teh device to do this calculation!\n");
	    exit(1);
	  }
	else 
	  {
	    printf("Hurrah! This device supports enough threads to do this calculation\n");
	  }
      }

  }// loop over devices
  
  return threadsPerBlock;
}
